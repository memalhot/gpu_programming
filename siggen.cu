#include "hip/hip_runtime.h"
// three matrixes, two rows in each matrix, three (columns) elements in each row
// int arr1[3][2][3] = {
//     {
//         {1, 2, 3},
//         {4, 5, 6}
//     },
//     {
//        {7, 8, 9},
//        {10, 11, 12}
//     },
//     {
//                {13, 14, 15},
//                {16, 17, 18}
//     }
// };


// int arr2[3][2][3] = {
//     {
//         {1, 2, 3},
//         {4, 5, 6}
//     },
//     {
//         {7, 8, 9},
//         {10, 11, 12}
//     },
//     {
//                 {13, 14, 15},
//                 {16, 17, 18}
//     }
// };


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <inttypes.h>
#include <time.h>

#define CLOCK_SOURCE CLOCK_MONOTONIC
#define NSEC_IN_SECOND (1000000000)

typedef struct timespec ts_t;

__global__ void f_siggen(float* A, float* B, float* C, int rows, int cols) {
    
    //int col = blockIdx.x * blockDim.x + threadIdx.x
    //int row = blockIdx.y * blockDim.y + threadIdx.y

    // i = row
    // j = col
    int i = threadIdx.y;
    int j = threadIdx.x;

    C[i * cols + j] = A[(i-1) * cols + j] + A[i * cols + j] + A[(i+1) * cols + j] - B[i * cols + j-2] - B[i * cols + j-1] - B[i * cols + j];
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <rows> <cols>\n", argv[0]);
        return 1;
    }

    int rows = atoi(argv[1]);
    int cols = atoi(argv[2]);
    int size = rows * cols * sizeof(float);

    // host
    float* h_A;
    float* h_B;
    float* h_C;

    // device
    float* d_A;
    float* d_B;
    float* d_C;

    // allocate for host
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    
    // allocate on device
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            int index = i * cols + j;

            h_A[index]=(float) ((i+j)%100)/2.0;
            h_B[index]=(float) 3.25*((i+j)%100);
        }
    }

    // hipMemcpy(dest,src,hipMemcpyDeviceToHost);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // generic thread blocks
    dim3 threadsPerBlock(rows, cols);

    // round up
    //int x = (cols + threadsPerBlock.x - 1) / threadsPerBlock.x;
    //int y = (rows + threadsPerBlock.y - 1) / threadsPerBlock.y;

    dim3 numBlocks(1, 1);


    static inline int ts_now(ts_t *now) {
    if (clock_gettime(CLOCK_SOURCE, now) == -1) {
        perror("clock_gettime");
        assert(0);
        return 0;
    }
    return 1;
    }

    f_siggen<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);
    hipError_t err = hipDeviceSynchronize();

    if (err != hipSuccess) {
        fprintf(stderr, "error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


    static inline uint64_t ts_diff(ts_t start, ts_t end)
        {
        uint64_t diff =
            ((end.tv_sec - start.tv_sec) * NSEC_IN_SECOND) +
            (end.tv_nsec - start.tv_nsec);
        return diff;
        } 

    //cleanup deivce
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset();

    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%f ", h_C[i * cols + j]);
        }
        printf("\n");
    }

    //cleanup host
    free(h_A);
    free(h_B);
    free(h_C);

    
}